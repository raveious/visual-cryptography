#include "hip/hip_runtime.h"

#include <stdio.h>
#include <fcntl.h>
#include <stdlib.h>
#include <sys/time.h>
#include <sys/stat.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
#include <stdint.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include "common.h"
#include "VisualCryptographyGPU.h"

#define ENCODE_TILE_SIZE 512
#define DECODE_TILE_SIZE 256

__constant__ level WhiteShare1[2][2] = {0,0,1,1};
__constant__ level WhiteShare2[2][2] = {0,0,1,1};

__constant__ level BlackShare1[2][2] = {0,0,1,1};
__constant__ level BlackShare2[2][2] = {1,1,0,0};

void CheckCUDAError(const char *msg) {
  hipError_t code =hipGetLastError();
  if(code!=hipSuccess)
  {
      fprintf(stderr,"Cuda Error: %s: %s.\n",msg,hipGetErrorString(code));
      exit(EXIT_FAILURE);
  }
}

__global__ void CodecKernel(level *pImage_d, level *pShare1_d, level *pShare2_d, int iWidth, int iHeight, int iCodecPath) {
  // ********************************************************************************
  // DO NOT CHANGE ANYTHING BEFORE THIS POINT in YOUR CODE                          *
  // Your CUDA Kernel should go here.                                               *
  // ********************************************************************************

  // ********************************************************************************
  // 1) If iCodecPath=ENCODE, the code performs encoding. In that case              *
  //    input image of size (iHeight x iWidth) pixels of (type level) stored at GPU *
  //    address pImage_d. The share images of size (2*iHeight x 2*iWidth) are stored*
  //    at GPU addresses pShare1_d and pShare2_d.                                   *
  // ********************************************************************************
  if (iCodecPath == ENCODE) {
    int offset = (iWidth * ENCODE_TILE_SIZE * blockIdx.y) + (blockDim.x * blockIdx.x) + threadIdx.x;
    int invert = 0;
    int share_offset = 0;
    int alt_share_offset = 0;
    int temp = 0;
    hiprandState_t state;
    hiprand_init(1234, offset, 0, &state);

    // loop over this threads segment
    for (int i = 0; i < ENCODE_TILE_SIZE * iWidth; i += iWidth) {
      invert = hiprand(&state) % 2;
      temp = i + offset;

      share_offset = (temp * 2) + ((temp / iWidth) * (iWidth * 2));
      alt_share_offset = share_offset + (iWidth * 2);

      // Select White or Black shares
      if (pImage_d[temp]) {
        pShare1_d[share_offset]         = WhiteShare1[0][0] ^ invert;
        pShare1_d[share_offset + 1]     = WhiteShare1[0][1] ^ invert;
        pShare1_d[alt_share_offset]     = WhiteShare1[1][0] ^ invert;
        pShare1_d[alt_share_offset + 1] = WhiteShare1[1][1] ^ invert;

        pShare2_d[share_offset]         = WhiteShare2[0][0] ^ invert;
        pShare2_d[share_offset + 1]     = WhiteShare2[0][1] ^ invert;
        pShare2_d[alt_share_offset]     = WhiteShare2[1][0] ^ invert;
        pShare2_d[alt_share_offset + 1] = WhiteShare2[1][1] ^ invert;
      } else {
        pShare1_d[share_offset]         = BlackShare1[0][0] ^ invert;
        pShare1_d[share_offset + 1]     = BlackShare1[0][1] ^ invert;
        pShare1_d[alt_share_offset]     = BlackShare1[1][0] ^ invert;
        pShare1_d[alt_share_offset + 1] = BlackShare1[1][1] ^ invert;

        pShare2_d[share_offset]         = BlackShare2[0][0] ^ invert;
        pShare2_d[share_offset + 1]     = BlackShare2[0][1] ^ invert;
        pShare2_d[alt_share_offset]     = BlackShare2[1][0] ^ invert;
        pShare2_d[alt_share_offset + 1] = BlackShare2[1][1] ^ invert;
      }
    }
  }
  // ********************************************************************************
  // 2) If iCodecPath=DECODE, the code performs decoding. In that case              *
  //    output image of size (iHeight x iWidth) pixels of (type level) is stored    *
  //    at GPU address pImage_d. The share images of size (iHeight x iWidth) are    *
  //    stored at GPU addresses pShare1_d and pShare2_d.                            *
  // ********************************************************************************
  else {
    int offset = (iWidth * DECODE_TILE_SIZE * blockIdx.y) + (blockDim.x * blockIdx.x) + threadIdx.x;
    int temp = 0;

    for (int i = 0; i < DECODE_TILE_SIZE * iWidth; i += iWidth) {
      temp = offset + i;
      // Safety stuff so threads don't run off the end
      if (temp < iHeight * iWidth) {
        // Take logical and of all the pixels to generate defined response
        pImage_d[temp] = pShare1_d[temp] & pShare2_d[temp];
      }

			// NOTE: The suppied solution uses logical NOR operation instead of
			// logical AND, but this doesn't yield the same result as what the
			// project description describes
    }
  }
  // ********************************************************************************
  // Your CUDA code ends here.                                                      *
  // DO NOT CHANGE ANYTHING AFTER THIS POINT in YOUR CODE                           *
  // ********************************************************************************
}

void VCEncoderGPU(ImageData *pcShare1, ImageData *pcShare2, ImageData *pcImageData, TimeRecord *pTR)
{
 	level *pShare1, *pShare2;
	level *pShare1_d, *pShare2_d;
	level *pImage_d;
	struct timeval start,stop;
	int blockSizeX, blockSizeY, gridSizeX, gridSizeY;

	printf("GPU Encoding... \n");

	//----------Fill in shares----------//
	pcShare1->iWidth  = 2 * pcImageData->iWidth;
	pcShare1->iHeight = 2 * pcImageData->iHeight;
	pcShare2->iWidth  = 2 * pcImageData->iWidth;
	pcShare2->iHeight = 2 * pcImageData->iHeight;
    hipDeviceReset();
	//Fill in file header
	memcpy(&(pcShare1->cBmpFH), &(pcImageData->cBmpFH), sizeof(BitMapFileHeader));
	memcpy(&(pcShare2->cBmpFH), &(pcImageData->cBmpFH), sizeof(BitMapFileHeader));
	//Fill in info header
	memcpy(&(pcShare1->cBmpIH), &(pcImageData->cBmpIH), sizeof(BitMapInfoHeader));
	memcpy(&(pcShare2->cBmpIH), &(pcImageData->cBmpIH), sizeof(BitMapInfoHeader));
	//Fill in color table
	memcpy(pcShare1->cBmpImage, pcImageData->cBmpImage, 2*sizeof(BitMapImage));
	memcpy(pcShare2->cBmpImage, pcImageData->cBmpImage, 2*sizeof(BitMapImage));

	//----------Adjust shares----------//
	sprintf(pcShare1->imageName, "Share1G_%s",pcImageData->imageName); //adjust name
	sprintf(pcShare2->imageName, "Share2G_%s",pcImageData->imageName);
	pcShare1->cBmpIH.width  = pcShare1->iWidth; //adjust width
	pcShare2->cBmpIH.width  = pcShare2->iWidth;
	pcShare1->cBmpIH.height = pcShare1->iHeight;  //adjust height
	pcShare2->cBmpIH.height = pcShare2->iHeight;
	pcShare1->cBmpIH.biSizeImage = pcShare1->cBmpIH.height * (((pcShare1->cBmpIH.bitPix * pcShare1->cBmpIH.width + 31) / 32) * 4); //adjust image size
	pcShare2->cBmpIH.biSizeImage = pcShare2->cBmpIH.height * (((pcShare2->cBmpIH.bitPix * pcShare2->cBmpIH.width + 31) / 32) * 4); //adjust image size
	pcShare1->cBmpFH.bfSize = pcShare1->cBmpIH.biSizeImage + pcShare1->cBmpFH.bfOffBits;
	pcShare2->cBmpFH.bfSize = pcShare2->cBmpIH.biSizeImage + pcShare2->cBmpFH.bfOffBits;

	//------Generate shares pixels---------//
	pcShare1->imgData = (level *)malloc(4 * pcImageData->iHeight * pcImageData->iWidth * sizeof(level));
	pcShare2->imgData = (level *)malloc(4 * pcImageData->iHeight * pcImageData->iWidth * sizeof(level));
	pShare1 = pcShare1->imgData;
	pShare2 = pcShare2->imgData;

	gettimeofday(&start,0);

	//------GPU Memory Preparation-------//
	hipMalloc( (void**)&pImage_d, pcImageData->iHeight * pcImageData->iWidth * sizeof(level));
	CheckCUDAError("Original Image GPU Memory Allocation Failed");
	hipMalloc( (void**)&pShare1_d, 4 * pcImageData->iHeight * pcImageData->iWidth * sizeof(level));
	CheckCUDAError("Share1 GPU Memory Allocation Failed");
	hipMemset(pShare1_d, 0, 4 * pcImageData->iHeight * pcImageData->iWidth * sizeof(level));
	CheckCUDAError("Share1 Image GPU Memory Set Failed");
	hipMalloc( (void**)&pShare2_d, 4 * pcImageData->iHeight * pcImageData->iWidth * sizeof(level));
	CheckCUDAError("Share2 GPU Memory Allocation Failed");
	hipMemset(pShare2_d, 0, 4 * pcImageData->iHeight * pcImageData->iWidth * sizeof(level));
	CheckCUDAError("Share2 Image GPU Memory Set Failed");

	//-------Transfer orignal image-------//
	hipMemcpy(pImage_d, pcImageData->imgData, pcImageData->iHeight * pcImageData->iWidth * sizeof(level), hipMemcpyHostToDevice);
	CheckCUDAError("Copy Original Image to GPU Failed");

	gettimeofday(&stop,0);
	pTR->MemTransferTime += ((stop.tv_sec - start.tv_sec) * 1000000 + (stop.tv_usec - start.tv_usec)) / 1000;

	gettimeofday(&start,0);
	//-----GPU Kernel Launch-----//
  // ********************************************************************************
  // DO NOT CHANGE ANYTHING BEFORE THIS POINT in YOUR CODE                          *
  // Your CUDA block size and grid size parameters go in here.                      *
  // ********************************************************************************
	//Fill in here

  // Each block is going to have ENCODE_TILE_SIZE threads that will run in seq. mem. blocks,
  // then do the line underneath that for ENCODE_TILE_SIZE lines.
	blockSizeX = ENCODE_TILE_SIZE;
  blockSizeY = 1;

  // Note: This means that the smallest image that can be decoded is
  // ENCODE_TILE_SIZE x ENCODE_TILE_SIZE
	gridSizeX  = pcImageData->iWidth / ENCODE_TILE_SIZE;
	gridSizeY  = pcImageData->iHeight / ENCODE_TILE_SIZE;

  if (gridSizeX < 1)
    gridSizeX = 1;

  if (gridSizeY < 1)
    gridSizeY = 1;
  // ********************************************************************************
  // End of CUDA block size and grid size parameters                                *
  // DO NOT CHANGE ANYTHING AFTER THIS POINT in YOUR CODE                           *
  // ********************************************************************************

	printf("|--Block Config: %d x %d\n",blockSizeX,blockSizeY);
	printf("|--Grid  Config: %d x %d\n",gridSizeX,gridSizeY);
	dim3 blocksInGrid(gridSizeX,gridSizeY);
	dim3 threadsInBlock(blockSizeX,blockSizeY);
	CodecKernel<<<blocksInGrid, threadsInBlock>>>(pImage_d,pShare1_d,pShare2_d,pcImageData->iWidth, pcImageData->iHeight, ENCODE);
	hipDeviceSynchronize();
	CheckCUDAError("Encryption Kernel Failed");

	gettimeofday(&stop,0);
	pTR->EncryptionTime += ((stop.tv_sec - start.tv_sec) * 1000000 + (stop.tv_usec - start.tv_usec)) / 1000;

	gettimeofday(&start,0);

	//------Transfer back shares------//
	hipMemcpy(pShare1,pShare1_d, 4 * pcImageData->iHeight * pcImageData->iWidth * sizeof(level), hipMemcpyDeviceToHost);
	CheckCUDAError("Copy Share1 to CPU Failed");
	hipMemcpy(pShare2,pShare2_d, 4 * pcImageData->iHeight * pcImageData->iWidth * sizeof(level), hipMemcpyDeviceToHost);
	CheckCUDAError("Copy Share2 to CPU Failed");

	gettimeofday(&stop,0);
	pTR->MemTransferTime += ((stop.tv_sec - start.tv_sec) * 1000000 + (stop.tv_usec - start.tv_usec)) / 1000;

	hipFree(pImage_d);
	hipFree(pShare1_d);
	hipFree(pShare2_d);
}

void VCDecoderGPU(ImageData *pcShare1, ImageData *pcShare2, char *pInputImageName, ImageData *pcImageData, TimeRecord *pTR)
{
 	level *pShare1, *pShare2;
	level *pShare1_d, *pShare2_d;
	level *pImage_d;
	struct timeval start,stop;
	int blockSizeX, blockSizeY, gridSizeX, gridSizeY;

	printf("GPU Decoding ...\n");
	pShare1 = pcShare1->imgData;
	pShare2 = pcShare2->imgData;
    hipDeviceReset();

	//------GPU Memory Preparation-------//
	memcpy(pcImageData, pcShare1, sizeof(ImageData));
	pcImageData->imgData = (level *)malloc(pcImageData->iHeight * pcImageData->iWidth * sizeof(level));
	sprintf(pcImageData->imageName, "ReconG_%s",pInputImageName);
	//Memory Allocation
	hipMalloc( (void**)&pImage_d, pcImageData->iHeight * pcImageData->iWidth * sizeof(level));
	CheckCUDAError("Reconstructed Image GPU Memory Allocation Failed");
	hipMemset(pImage_d, 0, pcImageData->iHeight * pcImageData->iWidth * sizeof(level));
	CheckCUDAError("Reconstructed Image GPU Memory Set Failed");
	hipMalloc( (void**)&pShare1_d, pcImageData->iHeight *  pcImageData->iWidth * sizeof(level));
	CheckCUDAError("Share1 GPU Memory Allocation Failed");
	hipMalloc( (void**)&pShare2_d, pcImageData->iHeight * pcImageData->iWidth * sizeof(level));
	CheckCUDAError("Share2 GPU Memory Allocation Failed");

	//Transfer shares
	gettimeofday(&start,0);
	hipMemcpy(pShare1_d, pShare1, pcImageData->iHeight * pcImageData->iWidth * sizeof(level), hipMemcpyHostToDevice);
	CheckCUDAError("Copy Share1 to GPU Failed");
	hipMemcpy(pShare2_d, pShare2, pcImageData->iHeight * pcImageData->iWidth * sizeof(level), hipMemcpyHostToDevice);
	CheckCUDAError("Copy Share2 to GPU Failed");
	gettimeofday(&stop,0);
	pTR->MemTransferTimeDecode += ((stop.tv_sec - start.tv_sec) * 1000000 + (stop.tv_usec - start.tv_usec)) / 1000;

	//----------GPU Kernel Launch----------//
	gettimeofday(&start,0);
  // ********************************************************************************
  // DO NOT CHANGE ANYTHING BEFORE THIS POINT in YOUR CODE                          *
  // Your CUDA block size and grid size parameters go in here.                      *
  // ********************************************************************************
	//Fill in here

  // Each block is going to have DECODE_TILE_SIZE threads that will run in seq. mem. blocks,
  // then do the line underneath that for DECODE_TILE_SIZE lines.
	blockSizeX = DECODE_TILE_SIZE;
  blockSizeY = 1;

  // Note: This means that the smallest image that can be decoded is
  // DECODE_TILE_SIZE x DECODE_TILE_SIZE
	gridSizeX  = pcImageData->iWidth / DECODE_TILE_SIZE;
	gridSizeY  = pcImageData->iHeight / DECODE_TILE_SIZE;

  if (gridSizeX < 1)
    gridSizeX = 1;

  if (gridSizeY < 1)
    gridSizeY = 1;
  // ********************************************************************************
  // End of CUDA block size and grid size parameters                                *
  // DO NOT CHANGE ANYTHING AFTER THIS POINT in YOUR CODE                           *
  // ********************************************************************************
	printf("|--Block Config: %d x %d\n",blockSizeX,blockSizeY);
	printf("|--Grid  Config: %d x %d\n",gridSizeX,gridSizeY);
	dim3 blocksInGrid(gridSizeX,gridSizeY);
	dim3 threadsInBlock(blockSizeX,blockSizeY);
	CodecKernel<<<blocksInGrid, threadsInBlock>>>(pImage_d,pShare1_d,pShare2_d,pcImageData->iWidth, pcImageData->iHeight, DECODE);
	hipDeviceSynchronize();
	CheckCUDAError("Decryption Kernel Failed");
	gettimeofday(&stop,0);
	pTR->DecodeTime += ((stop.tv_sec - start.tv_sec) * 1000000 + (stop.tv_usec - start.tv_usec)) / 1000;

	//------Transfer back reconstructed image------//
	gettimeofday(&start,0);
	hipMemcpy(pcImageData->imgData,pImage_d, pcImageData->iHeight * pcImageData->iWidth * sizeof(level), hipMemcpyDeviceToHost);
	CheckCUDAError("Copy Reconstructed image to CPU Failed");
	gettimeofday(&stop,0);
	pTR->MemTransferTimeDecode += ((stop.tv_sec - start.tv_sec) * 1000000 + (stop.tv_usec - start.tv_usec)) / 1000;

	//----------Free memory----------//
 	hipFree(pImage_d);
	hipFree(pShare1_d);
	hipFree(pShare2_d);

}
